#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdint.h>
#include <stdlib.h> 
#include <Windows.h>
#include <time.h>
#include "params.h"
#include "permutations.h"
#include "word.h"
#include "operations.h"
#include "ascon.h"

// HOST AEAD encryption
int crypto_aead_encrypt(uint8_t* c, uint64_t* clen, const uint8_t* m, uint64_t mlen, const uint8_t* ad, uint64_t adlen,
	const uint8_t* nsec, const uint8_t* npub, const uint8_t* k) {

	(void)nsec;
	/* set ciphertext size */
	*clen = mlen + CRYPTO_ABYTES;

	/* load key and nonce */
	const uint64_t K0 = LOADBYTES(k, 8);
	const uint64_t K1 = LOADBYTES(k + 8, 8);
	const uint64_t N0 = LOADBYTES(npub, 8);
	const uint64_t N1 = LOADBYTES(npub + 8, 8);

	/* initialize */
	state_t s;
	s.x0 = ASCON_128_IV;
	s.x1 = K0;
	s.x2 = K1;
	s.x3 = N0;
	s.x4 = N1;

	P12(&s);
	s.x3 ^= K0;
	s.x4 ^= K1;

	if (adlen) {
		/* full associated data blocks */
		while (adlen >= ASCON_128_RATE) {
			s.x0 ^= LOADBYTES(ad, 8);
			P6(&s);
			ad += ASCON_128_RATE;
			adlen -= ASCON_128_RATE;
		}
		s.x0 ^= LOADBYTES(ad, adlen);
		s.x0 ^= PAD(adlen);
		P6(&s);
	}
	/* domain separation */
	s.x4 ^= 1;

	/* full plaintext blocks */
	while (mlen >= ASCON_128_RATE) {
		s.x0 ^= LOADBYTES(m, 8);
		STOREBYTES(c, s.x0, 8);
		P6(&s);
		m += ASCON_128_RATE;
		c += ASCON_128_RATE;
		mlen -= ASCON_128_RATE;
	}
	/* final plaintext block */
	s.x0 ^= LOADBYTES(m, mlen);
	STOREBYTES(c, s.x0, mlen);
	s.x0 ^= PAD(mlen);
	c += mlen;

	/* finalize */
	s.x1 ^= K0;
	s.x2 ^= K1;
	P12(&s);
	s.x3 ^= K0;
	s.x4 ^= K1;

	/* set tag */
	STOREBYTES(c, s.x3, 8);
	STOREBYTES(c + 8, s.x4, 8);

	return 0;
}

__global__ void crypto_aead_encrypt_gpu(uint8_t* c, uint64_t* clen, const uint8_t* m, uint64_t mlen, const uint8_t* ad, uint64_t adlen,
	const uint8_t* nsec, const uint8_t* npub, const uint8_t* k) {
	(void)nsec;

	/* set ciphertext size */
	*clen = mlen + CRYPTO_ABYTES;

	uint32_t tid = threadIdx.x;	uint32_t bid = blockIdx.x;
	uint32_t idx_im = bid * blockDim.x * mlen + tid * mlen;								// message
	uint32_t idx_ia = bid * blockDim.x * adlen + tid * adlen;							// AD
	uint32_t idx_nk = bid * blockDim.x * CRYPTO_KEYBYTES + tid * CRYPTO_KEYBYTES;		// key and nonce read only 16
	uint32_t idx_out = bid * blockDim.x * (*clen) + tid * (*clen);						// cipher text length

	/* load key and nonce */
	const uint64_t K0 = LOADBYTESG(k + idx_nk, 8);
	const uint64_t K1 = LOADBYTESG(k + 8 + idx_nk, 8);
	const uint64_t N0 = LOADBYTESG(npub + idx_nk, 8);
	const uint64_t N1 = LOADBYTESG(npub + 8 + idx_nk, 8);

	/* initialize */
	state_t s;
	s.x0 = ASCON_128_IV;
	s.x1 = K0;
	s.x2 = K1;
	s.x3 = N0;
	s.x4 = N1;

	P12G(&s);
	s.x3 ^= K0;
	s.x4 ^= K1;

	if (adlen) {
		/* full associated data blocks */
		while (adlen >= ASCON_128_RATE) {
			s.x0 ^= LOADBYTESG(ad + idx_ia, 8);
			P6G(&s);
			ad += ASCON_128_RATE;
			adlen -= ASCON_128_RATE;
		}
		/* final associated data block */
		s.x0 ^= LOADBYTESG(ad + idx_ia, adlen);
		s.x0 ^= PAD(adlen);
		P6G(&s);
	}
	/* domain separation */
	s.x4 ^= 1;

	/* full plaintext blocks */
	while (mlen >= ASCON_128_RATE) {
		s.x0 ^= LOADBYTESG(m + idx_im, 8);
		STOREBYTESG(c + idx_out, s.x0, 8);
		P6G(&s);
		m += ASCON_128_RATE;
		c += ASCON_128_RATE;
		mlen -= ASCON_128_RATE;
	}
	/* final plaintext block */
	s.x0 ^= LOADBYTESG(m + idx_im, mlen);
	STOREBYTESG(c + idx_out, s.x0, mlen);
	s.x0 ^= PAD(mlen);
	c += mlen;

	/* finalize */
	s.x1 ^= K0;
	s.x2 ^= K1;
	P12G(&s);
	s.x3 ^= K0;
	s.x4 ^= K1;

	/* set tag */
	STOREBYTESG(c + idx_out, s.x3, 8);
	STOREBYTESG(c + idx_out + 8, s.x4, 8);
}

//GPU GlobalMem - read in col and write in row - transpose col
__global__ void crypto_aead_encrypt_gpu_rcwr_GpuTranspose(uint8_t* c, uint64_t* clen, const uint8_t* m, uint64_t mlen, const uint8_t* ad, uint64_t adlen,
	const uint8_t* nsec, const uint8_t* npub, const uint8_t* k) {
	(void)nsec;

	/* set ciphertext size */
	*clen = mlen + CRYPTO_ABYTES;

	/* Determine matrix index for each data*/
	uint32_t tkix = blockDim.x * blockIdx.x * CRYPTO_KEYBYTES + threadIdx.x;		//For Nonce and key - same because both 16 fixed
	uint32_t tkiy = blockDim.x * blockIdx.x * CRYPTO_KEYBYTES + threadIdx.y;
	uint32_t tmix = blockDim.x * blockIdx.x * mlen + threadIdx.y;					//for message with message len
	uint32_t tmiy = blockDim.x * blockIdx.x * mlen + threadIdx.y;
	uint32_t taix = blockDim.x * blockIdx.x * adlen + threadIdx.x;					//for additional data len
	uint32_t taiy = blockDim.x * blockIdx.x * adlen + threadIdx.y;
	uint32_t tcix = blockDim.x * blockIdx.x * (*clen) + threadIdx.x;				//for cipher text
	uint32_t tciy = blockDim.x * blockIdx.x * (*clen) + threadIdx.y;

	//read in col , write in row
	uint32_t tki = tkiy * CRYPTO_KEYBYTES + tkix; // access in rows - key & nonce
	uint32_t tko = tkix * CRYPTO_KEYBYTES + tkiy; // access in columns - key & nonce
	uint32_t tmi = tmiy * mlen + tmix; // access in rows - message 
	uint32_t tmo = tmix * mlen + tmiy; // access in columns - message 
	uint32_t tai = taiy * adlen + taix; // access in columns - ad 
	uint32_t tao = taix * adlen + taiy; // access in columns - ad 
	uint32_t tci = tciy * (*clen) + tcix; // access in row  - cipher

	/* load key and nonce */
	const uint64_t K0 = LOADBYTESG(k + tko, 8);
	const uint64_t K1 = LOADBYTESG(k + 8 + tko, 8);
	const uint64_t N0 = LOADBYTESG(npub + tko, 8);
	const uint64_t N1 = LOADBYTESG(npub + 8 + tko, 8);

	/* initialize */
	state_t s;
	s.x0 = ASCON_128_IV;
	s.x1 = K0;
	s.x2 = K1;
	s.x3 = N0;
	s.x4 = N1;

	P12G(&s);
	s.x3 ^= K0;
	s.x4 ^= K1;

	if (adlen) {
		/* full associated data blocks */
		while (adlen >= ASCON_128_RATE) {
			s.x0 ^= LOADBYTESG(ad + tao, 8);
			P6G(&s);
			ad += ASCON_128_RATE;
			adlen -= ASCON_128_RATE;
		}
		/* final associated data block */
		s.x0 ^= LOADBYTESG(ad + tao, adlen);
		s.x0 ^= PAD(adlen);
		P6G(&s);
	}
	/* domain separation */
	s.x4 ^= 1;

	/* full plaintext blocks */
	while (mlen >= ASCON_128_RATE) {
		s.x0 ^= LOADBYTESG(m + tmo, 8);
		STOREBYTESG(c + tci, s.x0, 8);
		P6G(&s);
		m += ASCON_128_RATE;
		c += ASCON_128_RATE;
		mlen -= ASCON_128_RATE;
	}
	/* final plaintext block */
	s.x0 ^= LOADBYTESG(m + tmo, mlen);
	STOREBYTESG(c + tci, s.x0, mlen);
	s.x0 ^= PAD(mlen);
	c += mlen;

	/* finalize */
	s.x1 ^= K0;
	s.x2 ^= K1;
	P12G(&s);
	s.x3 ^= K0;
	s.x4 ^= K1;

	/* set tag */
	STOREBYTESG(c + tci, s.x3, 8);
	STOREBYTESG(c + tci + 8, s.x4, 8);
}


//Optimised GPU transpose version
__global__ void crypto_aead_encrypt_gpu_global_Op_Trans(uint8_t* c, uint64_t* clen, const uint8_t* m, uint64_t mlen, const uint8_t* ad, uint64_t adlen,
	const uint8_t* nsec, const uint8_t* npub, const uint8_t* k) {
	(void)nsec;

	/* set ciphertext size */
	*clen = mlen + CRYPTO_ABYTES;

	/* Determine matrix index for each data*/
	uint32_t tkix = blockDim.x * blockIdx.x * CRYPTO_KEYBYTES + threadIdx.x;		//For Nonce and key - same because both 16 fixed
	uint32_t tkiy = blockDim.x * blockIdx.x * CRYPTO_KEYBYTES + threadIdx.y;
	uint32_t tmix = blockDim.x * blockIdx.x * mlen + threadIdx.y;					//for message with message len
	uint32_t tmiy = blockDim.x * blockIdx.x * mlen + threadIdx.y;
	uint32_t taix = blockDim.x * blockIdx.x * adlen + threadIdx.x;					//for additional data len
	uint32_t taiy = blockDim.x * blockIdx.x * adlen + threadIdx.y;
	uint32_t tcix = blockDim.x * blockIdx.x * (*clen) + threadIdx.x;				//for cipher text
	uint32_t tciy = blockDim.x * blockIdx.x * (*clen) + threadIdx.y;

	//read in col , write in row
	uint32_t tki = tkiy * CRYPTO_KEYBYTES + tkix; // access in rows - key & nonce
	uint32_t tko = tkix * CRYPTO_KEYBYTES + tkiy; // access in columns - key & nonce
	uint32_t tmi = tmiy * mlen + tmix; // access in rows - message 
	uint32_t tmo = tmix * mlen + tmiy; // access in columns - message 
	uint32_t tai = taiy * adlen + taix; // access in columns - ad 
	uint32_t tao = taix * adlen + taiy; // access in columns - ad 
	uint32_t tci = tciy * (*clen) + tcix; // access in row  - cipher

	/* load key and nonce */
	const uint64_t K0 = LOADBYTESG(k + tko, 8);
	const uint64_t K1 = LOADBYTESG(k + 8 + tko, 8);
	const uint64_t N0 = LOADBYTESG(npub + tko, 8);
	const uint64_t N1 = LOADBYTESG(npub + 8 + tko, 8);

	/* initialize */
	uint64_t x0, x1, x2, x3, x4;
	uint64_t t0, t1, t2, t3, t4;
	t0 = t1 = t2 = t3 = t4 = 0;

	x0 = ASCON_128_IV;
	x1 = K0;
	x2 = K1;
	x3 = N0;
	x4 = N1;

	P12_GO;
	x3 ^= K0;
	x4 ^= K1;

	if (adlen) {
		/* full associated data blocks */
		while (adlen >= ASCON_128_RATE) {
			x0 ^= LOADBYTESG(ad + tao, 8);
			P6_GO;
			ad += ASCON_128_RATE;
			adlen -= ASCON_128_RATE;
		}
		/* final associated data block */
		x0 ^= LOADBYTESG(ad + tao, adlen);
		x0 ^= PAD(adlen);
		P6_GO;
	}
	/* domain separation */
	x4 ^= 1;

	/* full plaintext blocks */
	while (mlen >= ASCON_128_RATE) {
		x0 ^= LOADBYTESG(m + tmo, 8);
		STOREBYTESG(c + tci, x0, 8);
		P6_GO;
		m += ASCON_128_RATE;
		c += ASCON_128_RATE;
		mlen -= ASCON_128_RATE;
	}
	/* final plaintext block */
	x0 ^= LOADBYTESG(m + tmo, mlen);
	STOREBYTESG(c + tci, x0, mlen);
	x0 ^= PAD(mlen);
	c += mlen;

	/* finalize */
	x1 ^= K0;
	x2 ^= K1;
	P12_GO;
	x3 ^= K0;
	x4 ^= K1;

	/* set tag */
	STOREBYTESG(c + tci, x3, 8);
	STOREBYTESG(c + tci + 8, x4, 8);
}

//GPU GlobalMem - transpose Col in Host & Unroll 4
__inline__ __device__ void encrypt_unroll4(uint8_t* c, uint64_t* clen, const uint8_t* m, uint64_t mlen, const uint8_t* ad, uint64_t adlen,
	const uint8_t* nsec, const uint8_t* npub, const uint8_t* k, uint32_t tko, uint32_t tao, uint32_t tmo, uint32_t tci) {

	const uint64_t K0 = LOADBYTESG(k + tko, 8);
	const uint64_t K1 = LOADBYTESG(k + 8 + tko, 8);
	const uint64_t N0 = LOADBYTESG(npub + tko, 8);
	const uint64_t N1 = LOADBYTESG(npub + 8 + tko, 8);

	/* initialize */
	state_t s;
	s.x0 = ASCON_128_IV;
	s.x1 = K0;
	s.x2 = K1;
	s.x3 = N0;
	s.x4 = N1;

	P12G(&s);
	s.x3 ^= K0;
	s.x4 ^= K1;

	if (adlen) {
		/* full associated data blocks */
		while (adlen >= ASCON_128_RATE) {
			s.x0 ^= LOADBYTESG(ad + tao, 8);
			P6G(&s);
			ad += ASCON_128_RATE;
			adlen -= ASCON_128_RATE;
		}
		/* final associated data block */
		s.x0 ^= LOADBYTESG(ad + tao, adlen);
		s.x0 ^= PAD(adlen);
		P6G(&s);
	}
	/* domain separation */
	s.x4 ^= 1;

	/* full plaintext blocks */
	while (mlen >= ASCON_128_RATE) {
		s.x0 ^= LOADBYTESG(m + tmo, 8);
		STOREBYTESG(c + tci, s.x0, 8);
		P6G(&s);
		m += ASCON_128_RATE;
		c += ASCON_128_RATE;
		mlen -= ASCON_128_RATE;
	}
	/* final plaintext block */
	s.x0 ^= LOADBYTESG(m + tmo, mlen);
	STOREBYTESG(c + tci, s.x0, mlen);
	s.x0 ^= PAD(mlen);
	c += mlen;

	/* finalize */
	s.x1 ^= K0;
	s.x2 ^= K1;
	P12G(&s);
	s.x3 ^= K0;
	s.x4 ^= K1;

	/* set tag */
	STOREBYTESG(c + tci, s.x3, 8);
	STOREBYTESG(c + tci + 8, s.x4, 8);
}

__global__ void crypto_aead_encrypt_gpu_rcwr_GPUTransposeUnroll4(uint8_t* c, uint64_t* clen, const uint8_t* m, uint64_t mlen, const uint8_t* ad, uint64_t adlen,
	const uint8_t* nsec, const uint8_t* npub, const uint8_t* k) {

	/* set ciphertext size */
	*clen = mlen + CRYPTO_ABYTES;

	/* Determine matrix index for each data*/
	uint32_t tkix = blockDim.x * blockIdx.x * CRYPTO_KEYBYTES + threadIdx.x;		//For Nonce and key - same because both 16 fixed
	uint32_t tkiy = blockDim.x * blockIdx.x * CRYPTO_KEYBYTES + threadIdx.y;
	uint32_t tmix = blockDim.x * blockIdx.x * mlen + threadIdx.y;					//for message with message len
	uint32_t tmiy = blockDim.x * blockIdx.x * mlen + threadIdx.y;
	uint32_t taix = blockDim.x * blockIdx.x * adlen + threadIdx.x;					//for additional data len
	uint32_t taiy = blockDim.x * blockIdx.x * adlen + threadIdx.y;
	uint32_t tcix = blockDim.x * blockIdx.x * (*clen) + threadIdx.x;				//for cipher text
	uint32_t tciy = blockDim.x * blockIdx.x * (*clen) + threadIdx.y;

	//read in col , write in row
	uint32_t tki = tkiy * CRYPTO_KEYBYTES + tkix; // access in rows - key & nonce
	uint32_t tko = tkix * CRYPTO_KEYBYTES + tkiy; // access in columns - key & nonce
	uint32_t tmi = tmiy * mlen + tmix; // access in rows - message 
	uint32_t tmo = tmix * mlen + tmiy; // access in columns - message 
	uint32_t tai = taiy * adlen + taix; // access in columns - ad 
	uint32_t tao = taix * adlen + taiy; // access in columns - ad 
	uint32_t tci = tciy * (*clen) + tcix; // access in row  - cipher

	//temporarily buffer
	uint8_t* kout = const_cast<uint8_t*>(k) + blockIdx.x * blockDim.x;
	uint8_t* nout = const_cast<uint8_t*>(npub) + blockIdx.x * blockDim.x;
	uint8_t* mout = const_cast<uint8_t*>(m) + blockIdx.x * blockDim.x;
	uint8_t* aout = const_cast<uint8_t*>(ad) + blockIdx.x * blockDim.x;

	kout[tko] = k[tki];													kout[tko + blockDim.x] = k[tki + blockDim.x];
	kout[tko + 2 * blockDim.x] = k[tki + 2 * blockDim.x];				kout[tko + 3 * blockDim.x] = k[tki + 3 * blockDim.x];

	nout[tko] = npub[tki];												nout[tko + blockDim.x] = npub[tki + blockDim.x];
	nout[tko + 2 * blockDim.x] = npub[tki + 2 * blockDim.x];			nout[tko + 3 * blockDim.x] = npub[tki + 3 * blockDim.x];

	mout[tmo] = m[tmi];													mout[tmo + blockDim.x] = m[tmi + blockDim.x];
	mout[tmo + 2 * blockDim.x] = m[tmi + 2 * blockDim.x];				mout[tmo + 3 * blockDim.x] = m[tmi + 3 * blockDim.x];

	aout[tao] = ad[tai];												aout[tao + blockDim.x] = ad[tai + blockDim.x];
	aout[tao + 2 * blockDim.x] = ad[tai + 2 * blockDim.x];				aout[tao + 3 * blockDim.x] = ad[tai + 3 * blockDim.x];

	__syncthreads();

	encrypt_unroll4(c, clen, mout, mlen, aout, adlen, nsec, nout, kout, tko, tao, tmo, tci);
	encrypt_unroll4(c, clen, mout, mlen, aout, adlen, nsec, nout, kout, tko + blockDim.x, tao + blockDim.x, tmo + blockDim.x, tci + blockDim.x);
	encrypt_unroll4(c, clen, mout, mlen, aout, adlen, nsec, nout, kout, tko + 2 * blockDim.x, tao + 2 * blockDim.x, tmo + 2 * blockDim.x, tci + 2 * blockDim.x);
	encrypt_unroll4(c, clen, mout, mlen, aout, adlen, nsec, nout, kout, tko + 3 * blockDim.x, tao + 3 * blockDim.x, tmo + 3 * blockDim.x, tci + 3 * blockDim.x);
}

int main()
{
	uint8_t* nonce, * key, * msg, * ad, * ct, * tag;
	uint64_t alen = ALEN;	// additional data length
	uint64_t mlen = MLEN;	// messege length
	uint64_t clen = MAX_CIPHER_LENGTH;	// cipher length
	int result = 0;

#ifdef WRITEFILE
	FILE* fpt;
	fpt = fopen("Ascon_Concurent.csv", "w");
	fprintf(fpt, "Version, Dimension, Threads, Latency, Memcpy H2D, Transpose, Execution Time, Memcpy D2H, AEAD/s (full latency), AEAD/s (exclude transpose)\n");
#endif

	//Memory allocation - HOST
	hipHostMalloc((void**)& key, BATCH * CRYPTO_KEYBYTES * sizeof(uint8_t));
	hipHostMalloc((void**)& nonce, BATCH * CRYPTO_NPUBBYTES * sizeof(uint8_t));
	hipHostMalloc((void**)& msg, BATCH * mlen * sizeof(uint8_t));
	hipHostMalloc((void**)& ad, BATCH * alen * sizeof(uint8_t));
	hipHostMalloc((void**)& ct, BATCH * clen * sizeof(uint8_t));
	hipHostMalloc((void**)& tag, BATCH * clen * sizeof(uint8_t));

	//Initialise key, nonce, message and additional data
	init_buffer('k', key, CRYPTO_KEYBYTES);
	init_buffer('n', nonce, CRYPTO_NPUBBYTES);
	init_buffer('m', msg, mlen);
	init_buffer('a', ad, alen);

	//CPU implementation
	LARGE_INTEGER frequency;
	LARGE_INTEGER t1, t2;
	double cpu_t = 0;

	QueryPerformanceFrequency(&frequency);
	QueryPerformanceCounter(&t1);
	for (int i = 0; i < BATCH; i++) {
		result |= crypto_aead_encrypt(OFFSET(ct, i, clen), &clen, OFFSET(msg, i, mlen), mlen, OFFSET(ad, i, alen), alen, 0,
			OFFSET(nonce, i, CRYPTO_NPUBBYTES), OFFSET(key, i, CRYPTO_KEYBYTES));
	}
	QueryPerformanceCounter(&t2);
	cpu_t += ((double)(t2.QuadPart - t1.QuadPart) * 1000.0 / (double)frequency.QuadPart);

	//Print Time
	printf("Version\tCKernel\tConfiguration\tMemcpyH2D\tMemcpyD2H\tLatency\t\tAEAD/s (full latency)\t AEAD/s (exclude transpose)\n\n");
#ifdef WRITEFILE
	fprintf(fpt, "%s, %.6f, %.6f, %.6f,%.6f, %.6f, %.6f, %.6f, %.6f, %.f, %.2f\n", "Host Op", 0, 0.0, cpu_t, 0.0, 0.0, cpu_t, 0.0, BATCH / (cpu_t / 1000), BATCH / (cpu_t / 1000));
#endif
	printf("Host \tSerial\t\t%.6f\t%.6f\t%.6f\t%.f\t%.f\n", 0.0, 0.0, cpu_t, BATCH / (cpu_t / 1000), BATCH / (cpu_t / 1000));

	//GPU implementation
	LARGE_INTEGER frequencyT;
	LARGE_INTEGER TS, TE;
	double trans = 0;
	uint8_t * key_out, *msg_out, *ad_out, *nonce_out;

	hipHostMalloc((void**)& key_out, BATCH * CRYPTO_KEYBYTES * sizeof(uint8_t));
	hipHostMalloc((void**)& msg_out, BATCH * mlen * sizeof(uint8_t));
	hipHostMalloc((void**)& ad_out, BATCH * alen * sizeof(uint8_t));
	hipHostMalloc((void**)& nonce_out, BATCH * CRYPTO_NPUBBYTES * sizeof(uint8_t));

	QueryPerformanceFrequency(&frequencyT);
	QueryPerformanceCounter(&TS);
	transposedata(key, key_out, BATCH, CRYPTO_KEYBYTES);
	transposedata(nonce, nonce_out, BATCH, CRYPTO_NPUBBYTES);
	transposedata(msg, msg_out, BATCH, mlen);
	transposedata(ad, ad_out, BATCH, alen);
	QueryPerformanceCounter(&TE);
	trans += (((double)(TE.QuadPart - TS.QuadPart) * 1000.0 / (double)frequencyT.QuadPart) / 100);

	uint8_t * d_n, *d_k, *d_a, *d_m, *d_c, *h_c;
	uint64_t * d_clen;
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	//Memory Allocation - Device
	hipHostMalloc((void**)& h_c, BATCH * clen * sizeof(uint8_t));
	hipMalloc((void**)& d_c, BATCH * clen * sizeof(uint8_t));
	hipMalloc((void**)& d_n, BATCH * CRYPTO_NPUBBYTES * sizeof(uint8_t));
	hipMalloc((void**)& d_k, BATCH * CRYPTO_KEYBYTES * sizeof(uint8_t));
	hipMalloc((void**)& d_m, BATCH * (uint64_t)mlen * sizeof(uint8_t));
	hipMalloc((void**)& d_a, BATCH * (uint64_t)alen * sizeof(uint8_t));
	hipHostMalloc((void**)& d_clen, sizeof(uint64_t));
	*d_clen = MAX_CIPHER_LENGTH;

	void (*kernel)(uint8_t*, uint64_t*, const uint8_t*, uint64_t, const uint8_t*, uint64_t, const uint8_t*, const uint8_t*, const uint8_t*);
	size_t size = BATCH * (*d_clen) * sizeof(uint8_t);

	hipStream_t GPUs2[2], GPUs4[4], GPUs5[5];
	hipStream_t * GPUstreams;

	for (int z = 2; z <= NSTREAM_SIZE; z++) {

		if (z != 3) {
			switch (z) {
			case 2: {GPUstreams = GPUs2; break; }
			case 4: {GPUstreams = GPUs4; break; }
			case 5: {GPUstreams = GPUs5; break; }
			}

			for (int a = 0; a < z; a++) {	//1 streams 8 bits
				CHECK(hipStreamCreate(&GPUstreams[a]));
			}

			//Determine data size
			int iBATCH = BATCH / z;
			size_t iKeysize = iBATCH * CRYPTO_KEYBYTES * sizeof(uint8_t);
			size_t iNoncesize = iBATCH * CRYPTO_NPUBBYTES * sizeof(uint8_t);
			size_t iMsgsize = iBATCH * (uint64_t)mlen * sizeof(uint8_t);
			size_t iAdsize = iBATCH * (uint64_t)alen * sizeof(uint8_t);
			size_t iCsize = iBATCH * (*d_clen) * sizeof(uint8_t);
			float memcpy_h2d;

			for (int i = 64; i < 1025; i *= 2) {

				//only thread 48 and above is executed since cipher have max 48 bytes to write.

				float elapsed, memcpy_d2h, total;

				for (int a = 1; a <= 4; a++) {
					if (a == 1) {
						hipEventRecord(start, 0);
						for (int i = 0; i < z; ++i)
						{
							int ioffset = i * iBATCH;
							hipMemcpyAsync(&d_n[ioffset * CRYPTO_NPUBBYTES], &nonce[ioffset * CRYPTO_NPUBBYTES], iNoncesize, hipMemcpyHostToDevice, GPUstreams[i]);
							hipMemcpyAsync(&d_k[ioffset * CRYPTO_KEYBYTES], &key[ioffset * CRYPTO_KEYBYTES], iKeysize, hipMemcpyHostToDevice, GPUstreams[i]);
							hipMemcpyAsync(&d_m[ioffset * mlen], &msg[ioffset * mlen], iMsgsize, hipMemcpyHostToDevice, GPUstreams[i]);
							hipMemcpyAsync(&d_a[ioffset * alen], &ad[ioffset * alen], iAdsize, hipMemcpyHostToDevice, GPUstreams[i]);
						}
						hipEventRecord(stop, 0);
						hipEventSynchronize(stop);
						memcpy_h2d = 0.0f;
						hipEventElapsedTime(&memcpy_h2d, start, stop);
					}
					else if (a == 2) {
						hipEventRecord(start, 0);
						for (int i = 0; i < z; ++i)
						{
							int ioffset = i * iBATCH;
							hipMemcpyAsync(&d_n[ioffset * CRYPTO_NPUBBYTES], &nonce_out[ioffset * CRYPTO_NPUBBYTES], iNoncesize, hipMemcpyHostToDevice, GPUstreams[i]);
							hipMemcpyAsync(&d_k[ioffset * CRYPTO_KEYBYTES], &key_out[ioffset * CRYPTO_KEYBYTES], iKeysize, hipMemcpyHostToDevice, GPUstreams[i]);
							hipMemcpyAsync(&d_m[ioffset * mlen], &msg_out[ioffset * mlen], iMsgsize, hipMemcpyHostToDevice, GPUstreams[i]);
							hipMemcpyAsync(&d_a[ioffset * alen], &ad_out[ioffset * alen], iAdsize, hipMemcpyHostToDevice, GPUstreams[i]);
						}
						hipEventRecord(stop, 0);
						hipEventSynchronize(stop);
						memcpy_h2d = 0.0f;
						hipEventElapsedTime(&memcpy_h2d, start, stop);
					}

					//Configuration.
					dim3 threads(i);
					dim3 blocks(iBATCH / i);		

					if (a > 1) {
						threads.y = i;
						blocks.x = (iBATCH / (threads.x * threads.y));
						blocks.x = (blocks.x < 1) ? 1 : blocks.x; // at least 1 block
					}

					//Determine Kernel Type.
					kernel = ((a == 1) ? &crypto_aead_encrypt_gpu : ((a == 2) ? &crypto_aead_encrypt_gpu_rcwr_GpuTranspose : ((a == 3) ? &crypto_aead_encrypt_gpu_rcwr_GPUTransposeUnroll4 :
						&crypto_aead_encrypt_gpu_global_Op_Trans)));
					char* kernelName = ((a == 1) ? "GPU Unoptimised" : ((a == 2) ? "GPU Tran" : ((a == 3) ? "GPU TransU4" : "GPU OpTrans ")));

					//Kernel execution
					memset(h_c, 0, BATCH * clen * sizeof(uint8_t));
					hipEventRecord(start);
					for (int i = 0; i < z; ++i) {
						int ioffset = i * iBATCH;
						kernel << <blocks, threads, 0, GPUstreams[i] >> > (&d_c[ioffset * clen], d_clen, &d_m[ioffset * mlen], mlen, &d_a[ioffset * alen], alen, 0,
							&d_n[ioffset * CRYPTO_NPUBBYTES], &d_k[ioffset * CRYPTO_KEYBYTES]);
					}
					hipEventRecord(stop);
					hipEventSynchronize(stop);
					hipEventElapsedTime(&elapsed, start, stop);

					//Memory Copy from D2H
					hipEventRecord(start, 0);
					for (int i = 0; i < z; ++i) {
						int ioffset = i * iBATCH;
						hipMemcpyAsync(&h_c[ioffset * clen], &d_c[ioffset * clen], iCsize, hipMemcpyDeviceToHost, GPUstreams[i]);
					}
					hipEventRecord(stop, 0);
					hipEventSynchronize(stop);
					hipEventElapsedTime(&memcpy_d2h, start, stop);

#ifdef CHECKRESULT
					checkResult(kernelName, ct, h_c, MAX_CIPHER_LENGTH);
#endif

					double Ttime = 0;
					if (a < 2)
						total = memcpy_h2d + elapsed + memcpy_d2h;
					else {
						total = memcpy_h2d + trans + elapsed + memcpy_d2h;
						Ttime = trans;
					}

					printf("%s\t %d\t %u \t\t%.6f\t%.6f\t%.6f  \t%.f \t%.f\n", kernelName, z, threads.x, memcpy_h2d,
						memcpy_d2h, total, BATCH / (total / 1000), BATCH / ((total - Ttime) / 1000));

#ifdef WRITEFILE
					fprintf(fpt, "%s,%d, %u, %.6f, %.6f, %.6f, %.6f,  %.6f, %.f, %.f\n", kernelName, z, threads.x, total,
						memcpy_h2d, Ttime, elapsed, memcpy_d2h, BATCH / (total / 1000), BATCH / ((total - Ttime) / 1000));
#endif

				}
				printf("\n");

			}
			printf("\n======================================================================================================================================================\n");

			for (int i = 0; i < z; i++)
				CHECK(hipStreamDestroy(GPUstreams[i]));
		}
	}

	//Free memory
	//Host memory
	hipFree(nonce);
	hipFree(key);
	hipFree(msg);
	hipFree(ad);
	hipFree(ct);
	hipFree(tag);

	//Device memory
	hipFree(d_n);
	hipFree(d_k);
	hipFree(d_a);
	hipFree(d_m);
	hipFree(d_c);
	hipFree(h_c);
	hipFree(d_clen);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipDeviceReset();
	return 0;
}