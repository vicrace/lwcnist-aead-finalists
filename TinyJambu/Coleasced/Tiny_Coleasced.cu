#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <stdio.h>
#include <Windows.h>
#include <math.h>
#include "params.h"
#include "Tiny128.h"
#include "operations.h"
//This K128 version

//Reference : Unop Ref = 0, Op Ref =1
int crypto_aead_encrypt(
	unsigned char* c, unsigned long long* clen,
	const unsigned char* m, unsigned long long mlen,
	const unsigned char* ad, unsigned long long adlen,
	const unsigned char* nsec,
	const unsigned char* npub,
	const unsigned char* k,
	unsigned int v
) {
	unsigned long long i;
	unsigned int j;
	unsigned char mac[8];
	unsigned int state[4];

	//initialization stage
	initialization_CPU(k, npub, state, v);

	//process the associated data   
	process_ad_CPU(k, ad, adlen, state, v);

	//process the plaintext    
	for (i = 0; i < (mlen >> 2); i++)
	{
		state[1] ^= FrameBitsPC;
		if (v == 0) state_update_Ref(state, k, NROUND2); else state_update_OpRef(state, k, NROUND2);
		state[3] ^= ((unsigned int*)m)[i];
		((unsigned int*)c)[i] = state[2] ^ ((unsigned int*)m)[i];
	}
	// if mlen is not a multiple of 4, we process the remaining bytes
	if ((mlen & 3) > 0)
	{
		state[1] ^= FrameBitsPC;
		if (v == 0) state_update_Ref(state, k, NROUND2);  else state_update_OpRef(state, k, NROUND2);
		for (j = 0; j < (mlen & 3); j++)
		{
			((unsigned char*)state)[12 + j] ^= m[(i << 2) + j];
			c[(i << 2) + j] = ((unsigned char*)state)[8 + j] ^ m[(i << 2) + j];
		}
		state[1] ^= mlen & 3;
	}

	//finalization stage, we assume that the tag length is 8 bytes
	state[1] ^= FrameBitsFinalization;
	if (v == 0) state_update_Ref(state, k, NROUND2); else state_update_OpRef(state, k, NROUND2);
	((unsigned int*)mac)[0] = state[2];

	state[1] ^= FrameBitsFinalization;
	if (v == 0) state_update_Ref(state, k, NROUND1);  else state_update_OpRef(state, k, NROUND1);
	((unsigned int*)mac)[1] = state[2];

	*clen = mlen + 8;
	for (j = 0; j < 8; j++) c[mlen + j] = mac[j];

	return 0;
}


__global__ void crypto_aead_encrypt_gpu_global_OpRef(
	uint8_t * c, uint64_t * clen,
	const uint8_t * m, uint64_t mlen,
	const uint8_t * ad, uint64_t adlen,
	const uint8_t * nsec,
	const uint8_t * npub,
	const uint8_t * k) {

	int tid = threadIdx.x, bid = blockIdx.x;
	if (bid * blockDim.x + tid < BATCH) {
		uint32_t offset_msg = bid * blockDim.x * mlen + tid * mlen;			// message
		uint32_t offset_ad = bid * blockDim.x * adlen + tid * adlen;			// AD
		uint32_t offset_key = bid * blockDim.x * CRYPTO_KEYBYTES + tid * CRYPTO_KEYBYTES; //key and nonce read only 16
		uint32_t offset_nonce = bid * blockDim.x * CRYPTO_NPUBBYTES + tid * CRYPTO_NPUBBYTES; //key and nonce read only 16
		uint32_t offset_ct = bid * blockDim.x * (MAX_CIPHER_LENGTH)+tid * (MAX_CIPHER_LENGTH);	//instead of crypto_abytes

		uint8_t* C = c + offset_ct;
		uint8_t* M = const_cast<uint8_t*>(m) + offset_msg;
		uint8_t* A = const_cast<uint8_t*>(ad) + offset_ad;
		uint8_t* N = const_cast<uint8_t*>(npub) + offset_nonce;
		uint8_t* K = const_cast<uint8_t*>(k) + offset_key;

		unsigned long long i;
		unsigned int j;
		unsigned char mac[8];
		unsigned int state[4];

		//initialization stage
		initialization_OpGPU(K, N, state);

		//process the associated data   
		process_ad_OpGPU(K, A, adlen, state);

		//process the plaintext    
		for (i = 0; i < (mlen >> 2); i++)
		{
			state[1] ^= FrameBitsPC;
			state_update_OpRefG(state, K, NROUND2);

			state[3] ^= ((unsigned int*)M)[i];
			((unsigned int*)C)[i] = state[2] ^ ((unsigned int*)M)[i];
		}
		// if mlen is not a multiple of 4, we process the remaining bytes
		if ((mlen & 3) > 0)
		{
			state[1] ^= FrameBitsPC;
			state_update_OpRefG(state, K, NROUND2);

			for (j = 0; j < (mlen & 3); j++)
			{
				((unsigned char*)state)[12 + j] ^= M[(i << 2) + j];
				C[(i << 2) + j] = ((unsigned char*)state)[8 + j] ^ M[(i << 2) + j];
			}
			state[1] ^= mlen & 3;
		}

		//finalization stage, we assume that the tag length is 8 bytes
		state[1] ^= FrameBitsFinalization;
		state_update_OpRefG(state, K, NROUND2);

		((unsigned int*)mac)[0] = state[2];

		state[1] ^= FrameBitsFinalization;
		state_update_OpRefG(state, K, NROUND1);

		((unsigned int*)mac)[1] = state[2];

		*clen = mlen + 8;
		for (j = 0; j < 8; j++) C[mlen + j] = mac[j];
	}
}


__global__ void crypto_aead_encrypt_gpu_global_Op(
	uint8_t * c, uint64_t * clen,
	const uint8_t * m, uint64_t mlen,
	const uint8_t * ad, uint64_t adlen,
	const uint8_t * nsec,
	const uint8_t * npub,
	const uint8_t * k) {

	int tid = threadIdx.x, bid = blockIdx.x;
	if (bid * blockDim.x + tid < BATCH) {
		uint32_t offset_msg = bid * blockDim.x * mlen + tid * mlen;			// message
		uint32_t offset_ad = bid * blockDim.x * adlen + tid * adlen;			// AD
		uint32_t offset_key = bid * blockDim.x * CRYPTO_KEYBYTES + tid * CRYPTO_KEYBYTES; //key and nonce read only 16
		uint32_t offset_nonce = bid * blockDim.x * CRYPTO_NPUBBYTES + tid * CRYPTO_NPUBBYTES; //key and nonce read only 16
		uint32_t offset_ct = bid * blockDim.x * (MAX_CIPHER_LENGTH)+tid * (MAX_CIPHER_LENGTH);	//instead of crypto_abytes

		uint8_t* C = c + offset_ct;
		uint8_t* M = const_cast<uint8_t*>(m) + offset_msg;
		uint8_t* A = const_cast<uint8_t*>(ad) + offset_ad;
		uint8_t* N = const_cast<uint8_t*>(npub) + offset_nonce;
		uint8_t* K = const_cast<uint8_t*>(k) + offset_key;

		unsigned long long i;
		unsigned int j;
		unsigned char mac[8];
		unsigned int state[4];

		//initialization stage
		initialization_GPU_Op(K, N, state);

		//process the associated data   
		process_ad_GPU_Op(K, A, adlen, state);

		//process the plaintext - unroll 2  
		for (i = 0; i < (mlen >> 2); i += 2)
		{
			state[1] ^= FrameBitsPC;
			state_update_OpRef_Register(state, K, NROUND2);

			state[3] ^= ((unsigned int*)M)[i];
			((unsigned int*)C)[i] = state[2] ^ ((unsigned int*)M)[i];

			//2nd time unroll
			state[1] ^= FrameBitsPC;
			state_update_OpRef_Register(state, K, NROUND2);

			state[3] ^= ((unsigned int*)M)[i + 1];
			((unsigned int*)C)[i + 1] = state[2] ^ ((unsigned int*)M)[i + 1];

		}

		// if mlen is not a multiple of 4, we process the remaining bytes
		if ((mlen & 3) > 0)
		{
			state[1] ^= FrameBitsPC;
			state_update_OpRef_Register(state, K, NROUND2);

			for (j = 0; j < (mlen & 3); j++)
			{
				((unsigned char*)state)[12 + j] ^= M[(i << 2) + j];
				C[(i << 2) + j] = ((unsigned char*)state)[8 + j] ^ M[(i << 2) + j];
			}
			state[1] ^= mlen & 3;
		}

		//finalization stage, we assume that the tag length is 8 bytes
		state[1] ^= FrameBitsFinalization;
		state_update_OpRef_Register(state, K, NROUND2);

		((unsigned int*)mac)[0] = state[2];

		state[1] ^= FrameBitsFinalization;
		state_update_OpRef_Register(state, K, NROUND1);

		((unsigned int*)mac)[1] = state[2];

		*clen = mlen + 8;
		for (j = 0; j < 8; j++) C[mlen + j] = mac[j];
	}
}

__global__ void crypto_aead_encrypt_gpu_global_Op_KeyInversion(
	uint8_t * c, uint64_t * clen,
	const uint8_t * m, uint64_t mlen,
	const uint8_t * ad, uint64_t adlen,
	const uint8_t * nsec,
	const uint8_t * npub,
	const uint8_t * k) {

	int tid = threadIdx.x, bid = blockIdx.x;
	if (bid * blockDim.x + tid < BATCH) {
		uint32_t offset_msg = bid * blockDim.x * mlen + tid * mlen;			// message
		uint32_t offset_ad = bid * blockDim.x * adlen + tid * adlen;			// AD
		uint32_t offset_key = bid * blockDim.x * CRYPTO_KEYBYTES + tid * CRYPTO_KEYBYTES; //key and nonce read only 16
		uint32_t offset_nonce = bid * blockDim.x * CRYPTO_NPUBBYTES + tid * CRYPTO_NPUBBYTES; //key and nonce read only 16
		uint32_t offset_ct = bid * blockDim.x * (MAX_CIPHER_LENGTH)+tid * (MAX_CIPHER_LENGTH);	//instead of crypto_abytes

		uint8_t* C = c + offset_ct;
		uint8_t* M = const_cast<uint8_t*>(m) + offset_msg;
		uint8_t* A = const_cast<uint8_t*>(ad) + offset_ad;
		uint8_t* N = const_cast<uint8_t*>(npub) + offset_nonce;
		uint8_t* key = const_cast<uint8_t*>(k) + offset_key;
		uint8_t* K = key;

		unsigned long long i;
		unsigned int j;
		unsigned char mac[8];
		unsigned int state[4];

		//keyInversion
		for (uint8_t i = 0; i < CRYPTO_KEYBYTES; i++) {
			K[i] = ~key[i];
		}

		//initialization stage
		initialization_OpGPU_Key(K, N, state);

		//process the associated data   
		process_ad_OpGPU_Key(K, A, adlen, state);

		//process the plaintext - unroll 2  
		for (i = 0; i < (mlen >> 2); i += 2)
		{
			state[1] ^= FrameBitsPC;
			state_update_Op_Key(state, K, NROUND2);

			state[3] ^= ((unsigned int*)M)[i];
			((unsigned int*)C)[i] = state[2] ^ ((unsigned int*)M)[i];

			//2nd time unroll
			state[1] ^= FrameBitsPC;
			state_update_Op_Key(state, K, NROUND2);

			state[3] ^= ((unsigned int*)M)[i + 1];
			((unsigned int*)C)[i + 1] = state[2] ^ ((unsigned int*)M)[i + 1];

		}

		// if mlen is not a multiple of 4, we process the remaining bytes
		if ((mlen & 3) > 0)
		{
			state[1] ^= FrameBitsPC;
			state_update_Op_Key(state, K, NROUND2);

			for (j = 0; j < (mlen & 3); j++)
			{
				((unsigned char*)state)[12 + j] ^= M[(i << 2) + j];
				C[(i << 2) + j] = ((unsigned char*)state)[8 + j] ^ M[(i << 2) + j];
			}
			state[1] ^= mlen & 3;
		}

		//finalization stage, we assume that the tag length is 8 bytes
		state[1] ^= FrameBitsFinalization;
		state_update_Op_Key(state, K, NROUND2);

		((unsigned int*)mac)[0] = state[2];

		state[1] ^= FrameBitsFinalization;
		state_update_Op_Key(state, K, NROUND1);

		((unsigned int*)mac)[1] = state[2];

		*clen = mlen + 8;
		for (j = 0; j < 8; j++) C[mlen + j] = mac[j];
	}
}

__global__ void crypto_aead_encrypt_gpu_global_OpRef_Trans(
	uint8_t * c, uint64_t * clen,
	const uint8_t * m, uint64_t mlen,
	const uint8_t * ad, uint64_t adlen,
	const uint8_t * nsec,
	const uint8_t * npub,
	const uint8_t * k) {

	if ((threadIdx.y * blockDim.x + threadIdx.x) < BATCH) {

		uint32_t tkix = blockDim.x * blockIdx.x * CRYPTO_KEYBYTES + threadIdx.x * CRYPTO_KEYBYTES;		//For Nonce and key - same because both 16 fixed
		uint32_t tkiy = blockDim.x * blockIdx.y * CRYPTO_KEYBYTES + (threadIdx.y * (blockDim.x * blockDim.x));
		uint32_t tnix = blockDim.x * blockIdx.x * CRYPTO_NPUBBYTES + threadIdx.x * CRYPTO_NPUBBYTES;		//For Nonce and key - same because both 16 fixed
		uint32_t tniy = blockDim.x * blockIdx.y * CRYPTO_NPUBBYTES + (threadIdx.y * (blockDim.x * blockDim.x));
		uint32_t tmix = blockDim.x * blockIdx.x * mlen + threadIdx.x * mlen;					//for message with message len
		uint32_t tmiy = blockDim.x * blockIdx.y * mlen + (threadIdx.y * (blockDim.x * blockDim.x)); // * 2
		uint32_t taix = blockDim.x * blockIdx.x * adlen + threadIdx.x * adlen;					//for additional data len
		uint32_t taiy = blockDim.x * blockIdx.y * adlen + (threadIdx.y * (blockDim.x * blockDim.x));
		uint32_t tcix = blockDim.x * blockIdx.x * (*clen) + threadIdx.x * (*clen);				//for cipher text
		uint32_t tciy = blockDim.x * blockIdx.y * (*clen) + (threadIdx.y * (blockDim.x * blockDim.x));

		uint32_t tk = tkiy * CRYPTO_KEYBYTES + tkix; // access in rows - key 
		uint32_t tn = tniy * CRYPTO_NPUBBYTES + tnix; // access in columns - nonce
		uint32_t tm = tmiy * mlen + tmix; // access in rows - message 
		uint32_t ta = taiy * adlen + taix; // access in columns - ad 
		uint32_t tc = tciy * (*clen) + tcix; // access in row  - cipher

		__syncthreads();
		uint8_t* C = c + tc;
		uint8_t* M = const_cast<uint8_t*>(m) + tm;
		uint8_t* A = const_cast<uint8_t*>(ad) + ta;
		uint8_t* N = const_cast<uint8_t*>(npub) + tn;
		uint8_t* K = const_cast<uint8_t*>(k) + tk;

		unsigned long long i;
		unsigned int j;
		unsigned char mac[8];
		unsigned int state[4];

		//initialization stage
		initialization_OpGPU(K, N, state);

		//process the associated data   
		process_ad_OpGPU(K, A, adlen, state);

		//process the plaintext    
		for (i = 0; i < (mlen >> 2); i++)
		{
			state[1] ^= FrameBitsPC;
			state_update_OpRefG(state, K, NROUND2);

			state[3] ^= ((unsigned int*)M)[i];
			((unsigned int*)C)[i] = state[2] ^ ((unsigned int*)M)[i];
		}
		// if mlen is not a multiple of 4, we process the remaining bytes
		if ((mlen & 3) > 0)
		{
			state[1] ^= FrameBitsPC;
			state_update_OpRefG(state, K, NROUND2);

#pragma unroll
			for (j = 0; j < (mlen & 3); j++)
			{
				((unsigned char*)state)[12 + j] ^= M[(i << 2) + j];
				C[(i << 2) + j] = ((unsigned char*)state)[8 + j] ^ M[(i << 2) + j];
			}
			state[1] ^= mlen & 3;
		}

		//finalization stage, we assume that the tag length is 8 bytes
		state[1] ^= FrameBitsFinalization;
		state_update_OpRefG(state, K, NROUND2);

		((unsigned int*)mac)[0] = state[2];

		state[1] ^= FrameBitsFinalization;
		state_update_OpRefG(state, K, NROUND1);

		((unsigned int*)mac)[1] = state[2];

		*clen = mlen + 8;
		for (j = 0; j < 8; j++) C[mlen + j] = mac[j];
	}
}


__global__ void crypto_aead_encrypt_gpu_global_Op_Trans(
	uint8_t * c, uint64_t * clen,
	const uint8_t * m, uint64_t mlen,
	const uint8_t * ad, uint64_t adlen,
	const uint8_t * nsec,
	const uint8_t * npub,
	const uint8_t * k) {

	if ((threadIdx.y * blockDim.x + threadIdx.x) < BATCH) {

		uint32_t tkix = blockDim.x * blockIdx.x * CRYPTO_KEYBYTES + threadIdx.x * CRYPTO_KEYBYTES;		//For Nonce and key - same because both 16 fixed
		uint32_t tkiy = blockDim.x * blockIdx.y * CRYPTO_KEYBYTES + (threadIdx.y * (blockDim.x * blockDim.x));
		uint32_t tnix = blockDim.x * blockIdx.x * CRYPTO_NPUBBYTES + threadIdx.x * CRYPTO_NPUBBYTES;		//For Nonce and key - same because both 16 fixed
		uint32_t tniy = blockDim.x * blockIdx.y * CRYPTO_NPUBBYTES + (threadIdx.y * (blockDim.x * blockDim.x));
		uint32_t tmix = blockDim.x * blockIdx.x * mlen + threadIdx.x * mlen;					//for message with message len
		uint32_t tmiy = blockDim.x * blockIdx.y * mlen + (threadIdx.y * (blockDim.x * blockDim.x)); // * 2
		uint32_t taix = blockDim.x * blockIdx.x * adlen + threadIdx.x * adlen;					//for additional data len
		uint32_t taiy = blockDim.x * blockIdx.y * adlen + (threadIdx.y * (blockDim.x * blockDim.x));
		uint32_t tcix = blockDim.x * blockIdx.x * (*clen) + threadIdx.x * (*clen);				//for cipher text
		uint32_t tciy = blockDim.x * blockIdx.y * (*clen) + (threadIdx.y * (blockDim.x * blockDim.x));

		////copy row
		uint32_t tk = tkiy * CRYPTO_KEYBYTES + tkix; // access in rows - key 
		uint32_t tn = tniy * CRYPTO_NPUBBYTES + tnix; // access in columns - nonce
		uint32_t tm = tmiy * mlen + tmix; // access in rows - message 
		uint32_t ta = taiy * adlen + taix; // access in columns - ad 
		uint32_t tc = tciy * (*clen) + tcix; // access in row  - cipher

		uint8_t* C = c + tc;
		uint8_t* M = const_cast<uint8_t*>(m) + tm;
		uint8_t* A = const_cast<uint8_t*>(ad) + ta;
		uint8_t* N = const_cast<uint8_t*>(npub) + tn;
		uint8_t* K = const_cast<uint8_t*>(k) + tk;

		unsigned long long i;
		unsigned int j;
		unsigned char mac[8];
		unsigned int state[4];

		//initialization stage
		initialization_GPU_Op_Trans(K, N, state);

		//process the associated data   
		process_ad_GPU_Op_Trans(K, A, adlen, state);

		//process the plaintext - unroll 2  
		for (i = 0; i < (mlen >> 2); i += 2)
		{
			state[1] ^= FrameBitsPC;
			state_update_OpRef_Register_Trans(state, K, NROUND2);

			state[3] ^= ((unsigned int*)M)[i];
			((unsigned int*)C)[i] = state[2] ^ ((unsigned int*)M)[i];

			//2nd time unroll
			state[1] ^= FrameBitsPC;
			state_update_OpRef_Register_Trans(state, K, NROUND2);

			state[3] ^= ((unsigned int*)M)[i + 1];
			((unsigned int*)C)[i + 1] = state[2] ^ ((unsigned int*)M)[i + 1];
		}

		// if mlen is not a multiple of 4, we process the remaining bytes
		if ((mlen & 3) > 0)
		{
			state[1] ^= FrameBitsPC;
			state_update_OpRef_Register_Trans(state, K, NROUND2);

			for (j = 0; j < (mlen & 3); j++)
			{
				((unsigned char*)state)[12 + j] ^= M[(i << 2) + j];
				C[(i << 2) + j] = ((unsigned char*)state)[8 + j] ^ M[(i << 2) + j];

			}
			state[1] ^= mlen & 3;
		}

		//finalization stage, we assume that the tag length is 8 bytes
		state[1] ^= FrameBitsFinalization;
		state_update_OpRef_Register_Trans(state, K, NROUND2);

		((unsigned int*)mac)[0] = state[2];

		state[1] ^= FrameBitsFinalization;
		state_update_OpRef_Register_Trans(state, K, NROUND1);

		((unsigned int*)mac)[1] = state[2];

		*clen = mlen + 8;
		for (j = 0; j < 8; j++) C[mlen + j] = mac[j];
	}
}

__global__ void crypto_aead_encrypt_gpu_global_Op_KeyInversion_Trans(
	uint8_t * c, uint64_t * clen,
	const uint8_t * m, uint64_t mlen,
	const uint8_t * ad, uint64_t adlen,
	const uint8_t * nsec,
	const uint8_t * npub,
	const uint8_t * k) {

	if ((threadIdx.y * blockDim.x + threadIdx.x) < BATCH) {
		/* Determine matrix index for each data*/
		uint32_t tkix = blockDim.x * blockIdx.x * CRYPTO_KEYBYTES + threadIdx.x;		//For Nonce and key - same because both 16 fixed
		uint32_t tkiy = blockDim.x * blockIdx.x * CRYPTO_KEYBYTES + threadIdx.y;
		uint32_t tnix = blockDim.x * blockIdx.x * CRYPTO_NPUBBYTES + threadIdx.x;		//For Nonce and key - same because both 16 fixed
		uint32_t tniy = blockDim.x * blockIdx.x * CRYPTO_NPUBBYTES + threadIdx.y;
		uint32_t tmix = blockDim.x * blockIdx.x * mlen + threadIdx.x;					//for message with message len
		uint32_t tmiy = blockDim.x * blockIdx.x * mlen + threadIdx.y;
		uint32_t taix = blockDim.x * blockIdx.x * adlen + threadIdx.x;					//for additional data len
		uint32_t taiy = blockDim.x * blockIdx.x * adlen + threadIdx.y;
		uint32_t tcix = blockDim.x * blockIdx.x * (*clen) + threadIdx.x;				//for cipher text
		uint32_t tciy = blockDim.x * blockIdx.x * (*clen) + threadIdx.y;

		//read in col , write in row
		uint32_t tki = tkiy * CRYPTO_KEYBYTES + tkix; // access in rows - key & nonce
		uint32_t tko = tkix * CRYPTO_KEYBYTES + tkiy; // access in columns - key & nonce
		uint32_t tni = tniy * CRYPTO_NPUBBYTES + tnix; // access in rows - key & nonce
		uint32_t tno = tnix * CRYPTO_NPUBBYTES + tniy; // access in columns - key & nonce
		uint32_t tmi = tmiy * mlen + tmix; // access in rows - message 
		uint32_t tmo = tmix * mlen + tmiy; // access in columns - message 
		uint32_t tai = taiy * adlen + taix; // access in columns - ad 
		uint32_t tao = taix * adlen + taiy; // access in columns - ad 
		uint32_t tci = tciy * (*clen) + tcix; // access in row  - cipher

		//temporarily buffer
		uint8_t* kout = const_cast<uint8_t*>(k) + blockIdx.x * blockDim.x;
		uint8_t* nout = const_cast<uint8_t*>(npub) + blockIdx.x * blockDim.x;
		uint8_t* mout = const_cast<uint8_t*>(m) + blockIdx.x * blockDim.x;
		uint8_t* aout = const_cast<uint8_t*>(ad) + blockIdx.x * blockDim.x;

		kout[tko] = k[tki]; // transpose from row to col for key
		nout[tno] = npub[tni]; //for nonce
		mout[tmo] = m[tmi]; //for message
		aout[tao] = ad[tai]; //for additional data

		__syncthreads();

		uint8_t* C = c + tci;
		uint8_t* M = mout + tmo;
		uint8_t* A = aout + tao;
		uint8_t* N = nout + tno;
		uint8_t* key = kout + tko;
		uint8_t* K = key;

		unsigned long long i;
		unsigned int j;
		unsigned char mac[8];
		unsigned int state[4];

		//keyInversion
		for (uint8_t i = 0; i < CRYPTO_KEYBYTES; i++) {
			K[i] = ~key[i];
		}

		//initialization stage
		initialization_OpGPU_Key(K, N, state);

		//process the associated data   
		process_ad_OpGPU_Key(K, A, adlen, state);

		//process the plaintext - unroll 2  
		for (i = 0; i < (mlen >> 2); i += 2)
		{
			state[1] ^= FrameBitsPC;
			state_update_Op_Key(state, K, NROUND2);

			state[3] ^= ((unsigned int*)M)[i];
			((unsigned int*)C)[i] = state[2] ^ ((unsigned int*)M)[i];

			//2nd time unroll
			state[1] ^= FrameBitsPC;
			state_update_Op_Key(state, K, NROUND2);

			state[3] ^= ((unsigned int*)M)[i + 1];
			((unsigned int*)C)[i + 1] = state[2] ^ ((unsigned int*)M)[i + 1];

		}

		// if mlen is not a multiple of 4, we process the remaining bytes
		if ((mlen & 3) > 0)
		{
			state[1] ^= FrameBitsPC;
			state_update_Op_Key(state, K, NROUND2);

			for (j = 0; j < (mlen & 3); j++)
			{
				((unsigned char*)state)[12 + j] ^= M[(i << 2) + j];
				C[(i << 2) + j] = ((unsigned char*)state)[8 + j] ^ M[(i << 2) + j];
			}
			state[1] ^= mlen & 3;
		}

		//finalization stage, we assume that the tag length is 8 bytes
		state[1] ^= FrameBitsFinalization;
		state_update_Op_Key(state, K, NROUND2);

		((unsigned int*)mac)[0] = state[2];

		state[1] ^= FrameBitsFinalization;
		state_update_Op_Key(state, K, NROUND1);

		((unsigned int*)mac)[1] = state[2];

		*clen = mlen + 8;
		for (j = 0; j < 8; j++) C[mlen + j] = mac[j];
	}
}

int main()
{

#ifdef WRITEFILE
	FILE* fpt;
	fpt = fopen("Tiny128_CA.csv", "w");
	fprintf(fpt, "Version, Threads, Latency, Memcpy H2D, Transpose, Execution Time, Memcpy D2H, AEAD/s (full latency), AEAD/s (exclude transpose)\n");
#endif

	uint8_t* nonce, * key, * msg, * ad, * ct, * ct_Op;
	uint64_t alen = ALEN;	// additional data length
	uint64_t mlen = MLEN;	// messege length
	uint64_t clen, clen2;	// cipher length
	LARGE_INTEGER frequency;
	LARGE_INTEGER t1, t2;
	double cpu_t = 0;

	hipHostMalloc((void**)& key, BATCH * CRYPTO_KEYBYTES * sizeof(uint8_t));
	hipHostMalloc((void**)& nonce, BATCH * CRYPTO_NPUBBYTES * sizeof(uint8_t));
	hipHostMalloc((void**)& msg, BATCH * mlen * sizeof(uint8_t));
	hipHostMalloc((void**)& ad, BATCH * alen * sizeof(uint8_t));
	hipHostMalloc((void**)& ct, BATCH * MAX_CIPHER_LENGTH * sizeof(uint8_t));
	hipHostMalloc((void**)& ct_Op, BATCH * MAX_CIPHER_LENGTH * sizeof(uint8_t));

	init_buffer('k', key, CRYPTO_KEYBYTES);
	init_buffer('n', nonce, CRYPTO_NPUBBYTES);
	init_buffer('m', msg, mlen);
	init_buffer('a', ad, alen);

	QueryPerformanceFrequency(&frequency);
	QueryPerformanceCounter(&t1);
	for (int i = 0; i < BATCH; i++) {
#ifdef PRINT
		print('m', msg + (i * mlen), mlen);
		printf(" -> ");
		print('a', ad + (i * alen), alen);
		print('k', key + (i * CRYPTO_KEYBYTES), CRYPTO_KEYBYTES);
		printf(" ");
		print('n', nonce + (i * CRYPTO_NPUBBYTES), CRYPTO_NPUBBYTES);
		printf(" ");
#endif

		int result = crypto_aead_encrypt(OFFSET(ct, i, clen), &clen, OFFSET(msg, i, mlen), mlen, OFFSET(ad, i, alen),
			alen, NULL, OFFSET(nonce, i, CRYPTO_NPUBBYTES), OFFSET(key, i, CRYPTO_KEYBYTES), Ref);
	}
	QueryPerformanceCounter(&t2);
	cpu_t += ((double)(t2.QuadPart - t1.QuadPart) * 1000.0 / (double)frequency.QuadPart);

#ifdef WRITEFILE
	fprintf(fpt, "%s, %.6f, %.6f, %.6f,%.6f, %.6f, %.6f, %.6f, %.6f, %.f, %.2f\n", "Host Ref", 0.0, cpu_t, 0.0, 0.0, cpu_t, 0.0, BATCH / (cpu_t / 1000), BATCH / (cpu_t / 1000));
#endif
	printf("Version\t\tConfiguration\tMemcpyH2D\tMemcpyD2H\tLatency\t\tAEAD/s (full latency)\t AEAD/s (exclude transpose)\n\n");
	printf("Host Ref\tSerial\t\t%.6f\t%.6f\t%.6f\t%.f\t\t\t%.f\n", 0.0, 0.0, cpu_t, BATCH / (cpu_t / 1000), BATCH / (cpu_t / 1000));


	//Op Ref
	QueryPerformanceFrequency(&frequency);
	QueryPerformanceCounter(&t1);
	for (int i = 0; i < BATCH; i++) {
		int result = crypto_aead_encrypt(OFFSET(ct_Op, i, clen2), &clen2, OFFSET(msg, i, mlen), mlen, OFFSET(ad, i, alen),
			alen, NULL, OFFSET(nonce, i, CRYPTO_NPUBBYTES), OFFSET(key, i, CRYPTO_KEYBYTES), Op);

#ifdef PRINTC
		print('cr', ct_Op + (i * clen2), clen2);
#endif
	}
	QueryPerformanceCounter(&t2);
	cpu_t = 0;
	cpu_t += ((double)(t2.QuadPart - t1.QuadPart) * 1000.0 / (double)frequency.QuadPart);

#ifdef WRITEFILE
	fprintf(fpt, "%s, %.6f, %.6f, %.6f,%.6f, %.6f, %.6f, %.6f, %.6f, %.f, %.2f\n", "Host Op", 0.0, cpu_t, 0.0, 0.0, cpu_t, 0.0, BATCH / (cpu_t / 1000), BATCH / (cpu_t / 1000));
#endif

	printf("Host Op\t\tSerial\t\t%.6f\t%.6f\t%.6f\t%.f\t\t\t%.f\n", 0.0, 0.0, cpu_t, BATCH / (cpu_t / 1000), BATCH / (cpu_t / 1000));


	//GPU implementation
	LARGE_INTEGER frequencyT;
	LARGE_INTEGER TS, TE;
	double trans = 0;
	uint8_t* key_out, * msg_out, * ad_out, * nonce_out;

	hipHostMalloc((void**)& key_out, BATCH * CRYPTO_KEYBYTES * sizeof(uint8_t));
	hipHostMalloc((void**)& msg_out, BATCH * mlen * sizeof(uint8_t));
	hipHostMalloc((void**)& ad_out, BATCH * alen * sizeof(uint8_t));
	hipHostMalloc((void**)& nonce_out, BATCH * CRYPTO_NPUBBYTES * sizeof(uint8_t));

	QueryPerformanceFrequency(&frequencyT);
	QueryPerformanceCounter(&TS);
	transposedata(key, key_out, BATCH, CRYPTO_KEYBYTES);
	transposedata(nonce, nonce_out, BATCH, CRYPTO_NPUBBYTES);
	transposedata(msg, msg_out, BATCH, mlen);
	transposedata(ad, ad_out, BATCH, alen);
	QueryPerformanceCounter(&TE);
	trans += (((double)(TE.QuadPart - TS.QuadPart) * 1000.0 / (double)frequencyT.QuadPart) / 100);

	uint8_t * d_n, *d_k, *d_a, *d_m, *d_c, *h_c;
	uint64_t * d_clen;
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	//Memory Allocation - Device
	hipHostMalloc((void**)& h_c, BATCH * MAX_CIPHER_LENGTH * sizeof(uint8_t));		//Host Cipher
	hipMalloc((void**)& d_c, BATCH * MAX_CIPHER_LENGTH * sizeof(uint8_t));			//Device Cipher
	hipMalloc((void**)& d_n, BATCH * CRYPTO_NPUBBYTES * sizeof(uint8_t));			//Nonce
	hipMalloc((void**)& d_k, BATCH * CRYPTO_KEYBYTES * sizeof(uint8_t));				//Key
	hipMalloc((void**)& d_m, BATCH * (uint64_t)mlen * sizeof(uint8_t));				//Message
	hipMalloc((void**)& d_a, BATCH * (uint64_t)alen * sizeof(uint8_t));				//Additional Data
	hipHostMalloc((void**)& d_clen, sizeof(uint64_t));

	//Memory initialisation
	memset(h_c, 0, BATCH * MAX_CIPHER_LENGTH * sizeof(uint8_t));
	hipMemset(d_c, 0, BATCH * MAX_CIPHER_LENGTH * sizeof(uint8_t));
	hipMemset(d_n, 0, BATCH * CRYPTO_NPUBBYTES * sizeof(uint8_t));
	hipMemset(d_k, 0, BATCH * CRYPTO_KEYBYTES * sizeof(uint8_t));
	hipMemset(d_m, 0, BATCH * (uint64_t)mlen * sizeof(uint8_t));
	hipMemset(d_a, 0, BATCH * (uint64_t)alen * sizeof(uint8_t));

	//Warm up kernel 
	crypto_aead_encrypt_gpu_global_OpRef << <BATCH / 1, 1 >> > (d_c, d_clen, d_m, mlen, d_a, alen, NULL, d_n, d_k);
	crypto_aead_encrypt_gpu_global_Op << <BATCH / 1, 1 >> > (d_c, d_clen, d_m, mlen, d_a, alen, NULL, d_n, d_k);
	crypto_aead_encrypt_gpu_global_Op_KeyInversion << <BATCH / 1, 1 >> > (d_c, d_clen, d_m, mlen, d_a, alen, NULL, d_n, d_k);

	void (*kernel)(uint8_t*, uint64_t*, const uint8_t*, uint64_t, const uint8_t*, uint64_t, const uint8_t*, const uint8_t*, const uint8_t*);
	size_t size = BATCH * (*d_clen) * sizeof(uint8_t);

	for (int i = 64; i < 1025; i *= 2) {
		float memcpy_h2d, elapsed, memcpy_d2h, total;

		for (int z = 1; z < 7; z++) {
			if (z == 1) { // for non-coleasced
				hipEventRecord(start, 0);
				CHECK(hipMemcpy(d_n, nonce, BATCH * CRYPTO_NPUBBYTES * sizeof(uint8_t), hipMemcpyHostToDevice));
				CHECK(hipMemcpy(d_k, key, BATCH * CRYPTO_KEYBYTES * sizeof(uint8_t), hipMemcpyHostToDevice));
				CHECK(hipMemcpy(d_m, msg, BATCH * (uint64_t)mlen * sizeof(uint8_t), hipMemcpyHostToDevice));
				CHECK(hipMemcpy(d_a, ad, BATCH * (uint64_t)alen * sizeof(uint8_t), hipMemcpyHostToDevice));
				hipEventRecord(stop, 0);
				hipEventSynchronize(stop);
				memcpy_h2d = 0.0f;
				hipEventElapsedTime(&memcpy_h2d, start, stop);
			}
			else if (z == 4) { // for coleasced
				hipEventRecord(start, 0);
				CHECK(hipMemcpy(d_n, nonce_out, BATCH * CRYPTO_NPUBBYTES * sizeof(uint8_t), hipMemcpyHostToDevice));
				CHECK(hipMemcpy(d_k, key_out, BATCH * CRYPTO_KEYBYTES * sizeof(uint8_t), hipMemcpyHostToDevice));
				CHECK(hipMemcpy(d_m, msg_out, BATCH * (uint64_t)mlen * sizeof(uint8_t), hipMemcpyHostToDevice));
				CHECK(hipMemcpy(d_a, ad_out, BATCH * (uint64_t)alen * sizeof(uint8_t), hipMemcpyHostToDevice));
				hipEventRecord(stop, 0);
				hipEventSynchronize(stop);
				memcpy_h2d = 0.0f;
				hipEventElapsedTime(&memcpy_h2d, start, stop);
			}

			dim3 threads(i);
			dim3 blocks(ceil((double)BATCH / (double)i));		//for unoptimised
			if (z > 3) {
				threads.y = i;
				double temp = (double)BATCH / ((double)threads.x * (double)threads.y);
				blocks.x = (temp < 1) ? 1 : ceil(temp); // at least 1 block
			}

			kernel = ((z == 1) ? &crypto_aead_encrypt_gpu_global_OpRef : ((z == 2) ? &crypto_aead_encrypt_gpu_global_Op :
				((z == 3) ? &crypto_aead_encrypt_gpu_global_Op_KeyInversion : ((z == 4) ? &crypto_aead_encrypt_gpu_global_OpRef_Trans :
				((z == 5) ? &crypto_aead_encrypt_gpu_global_Op_Trans :&crypto_aead_encrypt_gpu_global_Op_KeyInversion_Trans )))));

			char* kernelName = ((z == 1) ? "GPU Ref    " : ((z == 2) ? "GPU Op Reg " : ((z == 3) ? "GPU Op Inv  " : ((z == 4) ? "GPU Ref Trans" :
				((z == 5) ? "GPU Reg Trans " :  "GPU OpInv Trans" )))));


			//Kernel execution
			memset(h_c, 0, BATCH * MAX_CIPHER_LENGTH * sizeof(uint8_t));
			hipEventRecord(start, 0);
			kernel << <blocks, threads >> > (d_c, d_clen, d_m, mlen, d_a, alen, NULL, d_n, d_k);
			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);
			elapsed = 0.0f;
			hipEventElapsedTime(&elapsed, start, stop);

			//Memory Copy from D2H
			hipEventRecord(start, 0);
			hipMemcpy(h_c, d_c, BATCH * (*d_clen) * sizeof(uint8_t), hipMemcpyDeviceToHost);
			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);
			memcpy_d2h = 0.0f;
			hipEventElapsedTime(&memcpy_d2h, start, stop);
			checkResult(kernelName, ct, h_c, MAX_CIPHER_LENGTH);

			double Ttime = 0;
			if (z < 4)
				total = memcpy_h2d + elapsed + memcpy_d2h;
			else {
				total = memcpy_h2d + trans + elapsed + memcpy_d2h;
				Ttime = trans;
			}

			printf("%s\t %u \t\t%.6f\t%.6f\t%.6f  \t%.f \t\t%.f\n", kernelName, threads.x, memcpy_h2d,
				memcpy_d2h, total, BATCH / (total / 1000), BATCH / ((total - Ttime) / 1000));
#ifdef WRITEFILE
			fprintf(fpt, "%s, %u, %.6f, %.6f, %.6f, %.6f,  %.6f, %.f, %.f\n", kernelName, threads.x, total,
				memcpy_h2d, Ttime, elapsed, memcpy_d2h, BATCH / (total / 1000), BATCH / ((total - Ttime) / 1000));
#endif
		}
		printf("\n======================================================================================================================================================\n");
	}

	//Free Memory
	//Host memory
	hipFree(nonce);
	hipFree(key);
	hipFree(msg);
	hipFree(ad);
	hipFree(ct);
	hipFree(ct_Op);

	//Device memory
	hipFree(d_n);
	hipFree(d_k);
	hipFree(d_a);
	hipFree(d_m);
	hipFree(d_c);
	hipFree(h_c);
	hipFree(d_clen);

	hipEventDestroy(start);
	hipEventDestroy(stop);

#ifdef WRITEFILE
	fclose(fpt);
#endif

	hipDeviceReset();
}